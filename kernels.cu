#include "hip/hip_runtime.h"
#include "kernels.h"
#include <FreeImage.h>
#include <sstream>
#include "BatchLoad.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

/*
	Function name - distance
	arguments - x , y and z coodinates, counter to extract coordinates of control point
				the structure object which carries the coordinates
*/
__device__ uint16_t distance(int z,int y,int x,int counter, uint16_t* data)
{
	int cpx = (int)(data[3*counter+0]);
    int cpy = (int)data[3*counter+1];
    int cpz = (int)data[3*counter+2];
    int diff_x = powf((cpx-x),2);
    int diff_y = powf((cpy-y),2);
    int diff_z = powf((cpz-z),2);
    float displacement = (float) sqrtf(diff_x+diff_y+diff_z);
    return (uint16_t) displacement;
}

// sorting algorithm Although not the best technique
// This saves further function calls as in quicksort for partitioning
__device__ void selectionSort(uint16_t arr[], int high)
{
    int i, j, min, temp;
   	for (i = 0; i < high - 1; i++) {
      min = i;
      for (j = i + 1; j < high; j++)
      if (arr[j] < arr[min])
      min = j;
      temp = arr[i];
      arr[i] = arr[min];
      arr[min] = temp;
   }
}


__global__ void kernelcode(int l,int w,int h,uint16_t** GPUOutput,int nc,uint16_t* inputData,pc::Options* opts)
{
	int x =  threadIdx.x + blockIdx.x * blockDim.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int z = blockIdx.z;
    //const int arrsize = opts->numControlPoints;
    uint16_t dist[15]={};
    int index;
    // condition to check if more threads are spawned then execution should be restricted
	if ((x < l) && (y < w) && (z < h)) 
    {
		for(int numCtrlP = 0;numCtrlP<=opts->numControlPoints;numCtrlP++)
		{
			dist[numCtrlP] = distance(z,y,x,numCtrlP,inputData);
		}
		selectionSort(dist,15);
        index = (y * w * 3) + x * 3; 
        int red = (dist[1]/opts->dmax)*255;
        int blue = (dist[1]/opts->dmax)*255;
        int green = (dist[1]/opts->dmax)*255;
        //assigning color as per index
        GPUOutput[z][index + 0] = (int) red;
        GPUOutput[z][index + 1] = (int) green;
        GPUOutput[z][index + 2] = (int) blue;
	}
}

// there is no return values from cuda functions
// this is the only way to debug
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// extracting the coordinates of the control points
uint16_t* convert_to_string(int* size_out,pc::InputData* ob)
{
    *size_out = 6*ob->controlPositions.size();
    uint16_t* ptr = (uint16_t*) malloc(*size_out);
    
    for(int i = 0; i < ob->controlPositions.size(); i++)
    {
        int offset = i*3;
        ptr[offset+0] = (uint16_t) (ob->controlPositions[i].x);
        ptr[offset+1] = (uint16_t) (ob->controlPositions[i].y);
        ptr[offset+2] = (uint16_t) (ob->controlPositions[i].z);
    }

    return ptr;
}

void runGPUVariant(pc::InputData* obj,pc::Options* opts)
{
    // call dummyKernel1 ?!
    //std::cout<<"runGPUVariant called"<<std::endl;
    std::string outputFile = "outputgpu/plane_z";
    pc::Options* GPUoptions = 0;
    uint16_t* GPUinputData = 0;
    int cpSize = 0;
    uint16_t dimX;
    uint16_t dimY;
    uint16_t dimZ;
    int tcount;
    uint16_t* inputDataToString = convert_to_string(&cpSize,obj);
    hipMalloc((void**)&GPUoptions, 1000);
    hipMalloc((void**)&GPUinputData, cpSize);
    hipMemcpy(GPUoptions, opts, 1000, hipMemcpyHostToDevice);
    hipMemcpy(GPUinputData, inputDataToString, cpSize, hipMemcpyHostToDevice);
    //int hieght = opts->z;

    //dim3 grid(1,1,1);
    //dim3 block(2,2,2);
    uint16_t** OutputImg = (uint16_t**) malloc(opts->z * sizeof(uint16_t*));
    uint16_t** GPUOutputImg  = NULL;
    std::cout<<"runGPUVariant hipMemcpy"<<std::endl;
    //invalid argument error
    hipMalloc((void**)&GPUOutputImg, opts->z * sizeof(BYTE*));
    for(int i = 0; i < opts->z; i++){hipMalloc((void**)&OutputImg[i],opts->x*opts->y*3*sizeof(int));}
    gpuErrchk(hipMemcpy(GPUOutputImg, OutputImg, opts->z * sizeof(uint16_t*), hipMemcpyHostToDevice));
    
    struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	std::cout<<"using "<<properties.multiProcessorCount<<" multiprocessors"<<std::endl;
	std::cout<<"max threads per processor: "<<properties.maxThreadsPerMultiProcessor<<std::endl;
    std::cout<<"@@@ Launching Kernel @@@@"<<std::endl;
    if(opts->mode == pc::Mode::GPUVersion2)
    {
    	dimX = (opts->x / 10);
    	dimY = (opts->y / 10);
    	dimZ = opts->z;
    	const dim3 threadsPerBlock(10,10,1);
    	const dim3 blocksPerGrid(dimX,dimY,dimZ);
    
    	kernelcode <<< blocksPerGrid, threadsPerBlock >>> (opts->x,opts->y,opts->z,GPUOutputImg,opts->numControlPoints,GPUinputData,GPUoptions);
    	gpuErrchk(hipDeviceSynchronize());

	
    }
    else
    {
    	tcount = sqrt(properties.maxThreadsPerBlock);
    	dimX =  opts->x / tcount;
    	dimY = opts->y / tcount;
    	dimZ = opts->z;
    	const dim3 threadsPerBlock(tcount,tcount,1);
    	const dim3 blocksPerGrid(dimX,dimY,dimZ);
    	kernelcode <<< blocksPerGrid, threadsPerBlock >>> (opts->x,opts->y,opts->z,GPUOutputImg,opts->numControlPoints,GPUinputData,GPUoptions);
		gpuErrchk(hipDeviceSynchronize());

    }
	std::cout<<"@@@ CUDA Kernel processing @@@@"<<std::endl;
	hipError_t err = hipGetLastError();        // Get error code

   	if ( err != hipSuccess )
   	{
      printf("CUDA Error: %s\n", hipGetErrorString(err));
      exit(-1);
   	}
   	//idea is to keep z constant and calculate distance of a pixel with remaining control points
    for(int z =0;z<opts->z;z++)
    {
    	//std::cout<<"@@@ Screening z axis @@@@"<<std::endl;
    		
    	FIBITMAP* gpuOutputImage = FreeImage_AllocateT(FIT_BITMAP, opts->x, opts->y, 24);
        BYTE* CPUImageptr = FreeImage_GetBits(gpuOutputImage);

        // Copies calculated image from GPU memory to the FIBITMAP Object and clean up
        gpuErrchk(hipMemcpy(CPUImageptr, OutputImg[z], opts->x * opts->y * sizeof(BYTE) * 3, hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(OutputImg[z]));
        // will be needed for ffmpeg
        std::string name = "outputgpu/plane_z";
	    if(z<10)
	    {
	        name = name+"00"+std::to_string(z)+".png";
	    }
	    else if(z<100)
	    {
	        name = name+"0"+std::to_string(z)+".png";
	    }
	    else
	    {
	        name = name+std::to_string(z)+".png";
	    }
	    std::cout<<" Screened z axis @@@@"<<z<<std::endl;
    	bool returnvalue = GenericWriter(gpuOutputImage, name.c_str(), PNG_DEFAULT);
    	std::cout<<" Returned value is "<<returnvalue<<std::endl;
    	FreeImage_Unload(gpuOutputImage);


    }
    err = hipGetLastError();        // Get error code

   	if ( err != hipSuccess )
   	{
      printf("CUDA Error: %s\n", hipGetErrorString(err));
      exit(-1);
   	}
}
